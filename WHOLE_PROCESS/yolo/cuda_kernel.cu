#include "hip/hip_runtime.h"
//
// Created by Jarry_Goon on 2023-09-16.
//

#include <opencv2/opencv.hpp>

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#define BLOCK   1024
#define BLOCK2D dim3(32, 32)
#define BLOCK3D dim3(16, 16, 4)

#define GRID(SIZE) (((SIZE) + BLOCK - 1) / BLOCK)
#define GRID2D(X, Y) dim3((((X) + BLOCK2D.x - 1) / BLOCK2D.x), \
                          (((Y) + BLOCK2D.y - 1) / BLOCK2D.y))
#define GRID3D(X, Y, Z) dim3((((X) + BLOCK3D.x - 1) / BLOCK3D.x), \
                             (((Y) + BLOCK3D.y - 1) / BLOCK3D.y), \
                             (((Z) + BLOCK3D.z - 1) / BLOCK3D.z))

__global__
void half2float(const half* src, float* dst, int size)
{
	int array_idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (array_idx < size)
		dst[ array_idx ] = __half2float(src[ array_idx ]);
}

__global__
void float2half(const float* src, half* dst, int size)
{
	int array_idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (array_idx < size)
	{
		dst[ array_idx ] = __float2half_rn(src[ array_idx ]);
	}
}

__global__
void img2blob_kernel(const float* _img, half* blob, int _rows, int _cols, int _channels)
{
	int row     = blockIdx.x * blockDim.x + threadIdx.x;
	int col     = blockIdx.y * blockDim.y + threadIdx.y;
	int channel = blockIdx.z * blockDim.z + threadIdx.z;
	
	if (row < _rows && col < _cols && channel < _channels)
	{
		blob[ channel * _rows * _cols + row * _cols + col ] = __float2half_rn(
				_img[ channel + col * _channels + row * _channels * _cols ] / 255.f);
	}
}

__global__
void transpose_kernel(const half* src, float* dst, int _num_classes)
{
	int row = 4 + _num_classes + 32;
	
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x < row && y < 8400)
		dst[ x + row * y ] = __half2float(src[ 8400 * x + y ]);
}

__global__
void find_max_score_kernel(const float* _box_array, float* _scores, int* _ids,
                           int _num_class, float _threshold, int* _size)
{
	int rows = 4 + _num_class + 32;
	int col  = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (col < 8400)
	{
		float max = _box_array[ rows * col + 4 ];
		float temp;
		
		for (int i = 5; i < _num_class + 4; i++)
		{
			temp = _box_array[ rows * col + i ];
			if (temp > max)
			{
				max = temp;
				_ids[ col ] = i;
			}
		}
		
		if (max > 0.000001) printf("%f\n", max);
		
		if (max > _threshold)
		{
			_scores[ col ] = max;
			atomicAdd(_size, 1);
		}
		else
			_scores[ col ] = -1.f;
	}
}

__global__
void score_check_kernel(const int* _ids, const float* _scores, int* _class_ids, int* _confirmed_idx)
{
	int idx = 0;
	
	for (int i = 0; i < 8400; i++)
	{
		if (_scores[ i ] == -1.f) continue;
		
		_confirmed_idx[ idx ] = i;
		_class_ids[ idx ]     = _ids[ i ];
		idx++;
	}
}

__global__
void predict_kernel(const int* _confirmed_idx, const float* _box_data, float* _predict_mask,
                    cv::Rect* boxes, int _size, int _box_rows, float _x_factor, float _y_factor, cv::Size _img_size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (idx < _size)
	{
		int conf_idx = _confirmed_idx[ idx ];
		
		float x = _box_data[ _box_rows * conf_idx ];
		float w = _box_data[ _box_rows * conf_idx + 2 ];
		float y = _box_data[ _box_rows * conf_idx + 1 ];
		float h = _box_data[ _box_rows * conf_idx + 3 ];
		
		int x_min = (int) (( x - 0.5 * w ) * _x_factor );
		int x_max = (int) (( x + 0.5 * w ) * _x_factor );
		int y_min = (int) (( y - 0.5 * h ) * _y_factor );
		int y_max = (int) (( y + 0.5 * h ) * _y_factor );
		
		if (x_min < 0) x_min                = 0;
		if (x_max > _img_size.width) x_max  = _img_size.width;
		if (y_min < 0) y_min                = 0;
		if (y_max > _img_size.height) y_max = _img_size.height;
		
		boxes[ idx ].x      = x_min;
		boxes[ idx ].y      = y_min;
		boxes[ idx ].width  = x_max - x_min;
		boxes[ idx ].height = y_max - y_min;
		
		for (int i = 0; i < 32; i++)
			_predict_mask[ 32 * conf_idx + i ] = _box_data[ _box_rows * ( conf_idx + 1 ) - 32 + i ];
	}
}
