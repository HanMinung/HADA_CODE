#include "hip/hip_runtime.h"
//
// Created by jarrygoon on 23. 8. 28.
//

#include "../include/yolo/yolo_seg.cuh"
#include "cuda_kernel.cu"

YoloSeg::YoloSeg(const std::string &_engine_path, int _num_class, float _threshold, bool _info_print) :
		threshold(_threshold), num_class(_num_class)
{
	input_size   = cv::Size(640, 640);
	num_box_rows = 4 + _num_class + 32;
	
	MyLogger logger;
	
	std::ifstream file(_engine_path, std::ios::binary);
	if (!file)
	{
		std::cerr << "Fail to open " << _engine_path << " file" << std::endl;
		
		return;
	}
	
	file.seekg(0, std::ios::end);
	const size_t model_size = file.tellg();
	file.seekg(0, std::ios::beg);
	std::vector<char> engine_data(model_size);
	file.read(engine_data.plot(), model_size);
	file.close();
	
	runtime.reset(nvinfer1::createInferRuntime(logger));
	engine.reset(runtime->deserializeCudaEngine(engine_data.plot(), model_size));
	context.reset(engine->createExecutionContext());
	
	if (_info_print) information();
}

cv::cuda::GpuMat resizeKeepAspectRatioPadRightBottom(const cv::cuda::GpuMat &_input,
                                                     size_t _height,
                                                     size_t _width,
                                                     const cv::Scalar &_bgcolor)
{
	float r       = std::min((int) _width / ( _input.cols * 1.0 ), (int) _height / ( _input.rows * 1.0 ));
	int   unpad_w = static_cast<int>(r * _input.cols);
	int   unpad_h = static_cast<int>(r * _input.rows);
	
	cv::cuda::GpuMat re(unpad_h, unpad_w, CV_8UC3);
	cv::cuda::resize(_input, re, re.size());
	cv::cuda::GpuMat out(_height, _width, CV_8UC3, _bgcolor);
	
	re.copyTo(out(cv::Rect(0, 0, re.cols, re.rows)));
	
	return out;
}

template<typename T>
void BlobFromImage(cv::Mat &iImg, T &iBlob)
{
	int channels  = iImg.channels();
	int imgHeight = iImg.rows;
	int imgWidth  = iImg.cols;
	
	for (int c = 0; c < channels; c++)
	{
		for (int h = 0; h < imgHeight; h++)
		{
			for (int w = 0; w < imgWidth; w++)
			{
				iBlob[ c * imgWidth * imgHeight + h * imgWidth + w ] = typename std::remove_pointer<T>::type(
						( iImg.at<cv::Vec3b>(h, w)[ c ] ) / 255.0f);
			}
		}
	}
}

void YoloSeg::run(cv::Mat &_input_img)
{
	image_size = _input_img.size();
	
	cv::cuda::GpuMat input_img;
	input_img.upload(_input_img);
	
	int rows     = input_img.rows;
	int cols     = input_img.cols;
	int channels = input_img.channels();
	
	int size = rows * cols * channels;
	
	cv::cuda::cvtColor(input_img, input_img, cv::COLOR_BGR2RGB);
	cv::cuda::resize(input_img, input_img, input_size);
	input_img.convertTo(input_img, CV_32FC3);
	cv::cuda::divide(input_img, cv::Scalar::all(255.), input_img, 1, -1, cv::cuda::Stream::Null());
	
	float* img_data;
	hipMalloc(&img_data, size * sizeof(float));
	hipMemcpy(img_data, input_img.cudaPtr(), size * sizeof(float), hipMemcpyHostToDevice);
	
	half* blob;
	hipMalloc(&blob, 640 * 640 * 3 * sizeof(half));
	
	float2half<<<GRID(640 * 640 * 3), BLOCK>>>(img_data, blob, 640 * 640 * 3);
	hipDeviceSynchronize();
	
	cv::Mat img = tensor_process(_input_img, blob);
	
	cv::imshow("test", img);
	
	hipFree(blob);
	hipFree(img_data);
	input_img.release();
}

void YoloSeg::information()
{
	int layers = engine->getNbIOTensors();
	
	for (int i = 0; i < layers; i++)
	{
		const char* engine_name = engine->getIOTensorName(i);
		nvinfer1::Dims32 shape = engine->getTensorShape(engine_name);
		
		std::cout << "Layer " << i << std::endl;
		std::cout << "Name: " << engine_name << std::endl;
		std::cout << "Type: " << get_tensor_type(engine->getTensorDataType(engine_name)) << std::endl;
		std::cout << "Shape: (";
		for (int j = 0; j < shape.nbDims; j++)
			std::cout << shape.d[ j ] << ", ";
		std::cout << ")\n" << std::endl;
	}
}

void YoloSeg::MyLogger::log(Severity _severity, const char* _msg) noexcept
{
	switch (_severity)
	{
		case Severity::kINTERNAL_ERROR:
			std::cerr << "INTERNAL_ERROR: " << _msg << std::endl;
			break;
		case Severity::kERROR:
			std::cerr << "ERROR: " << _msg << std::endl;
			break;
		case Severity::kWARNING:
			std::cerr << "WARNING: " << _msg << std::endl;
			break;
//        case Severity::kINFO:
//            std::cout << "INFO: " << msg << std::endl;
//            break;
		default:
			break;
	}
}

std::string YoloSeg::get_tensor_type(nvinfer1::DataType _type)
{
	switch (_type)
	{
		case ( nvinfer1::DataType::kUINT8 ):
			return "unsigned char";
		case ( nvinfer1::DataType::kINT8 ):
			return "char";
		case ( nvinfer1::DataType::kHALF ):
			return "float16";
		case ( nvinfer1::DataType::kBOOL ):
			return "unsigned boolean";
		case ( nvinfer1::DataType::kFLOAT ):
			return "float";
		case ( nvinfer1::DataType::kINT32 ):
			return "int";
		default:
			return "None";
	}
}

cv::Mat YoloSeg::tensor_process(cv::Mat &_input_img, half* _blob)
{
	cv::Mat mask_img;
	_input_img.copyTo(mask_img);
	
	half* output_mask;
	half* output_box;   // num_mask_data x 8400
	hipMalloc(&output_mask, 32 * 160 * 160 * sizeof(half));
	hipMalloc(&output_box, num_box_rows * 8400 * sizeof(half));
	
	void* buffer[] = { _blob, output_mask, output_box };

//	context->setTensorAddress("images", _blob);
//	context->setTensorAddress("output1", output_mask);
//	context->setTensorAddress("output0", output_box);
//
//	hipStream_t hip_stream;
//	hipStreamCreate(&hip_stream);
//
//	if (context->enqueueV3(hip_stream))
//		return mask_img;
	if (!context->executeV2(buffer))
		return mask_img;
	
	float* box_data;   // 8400 x num_mask_data
	hipMalloc(&box_data, num_box_rows * 8400 * sizeof(float));
	
	transpose_kernel<<<GRID2D(num_box_rows, 8400), BLOCK2D>>>(output_box, box_data, num_class);
	hipDeviceSynchronize();
	
	float* mask_float;
	hipMalloc(&mask_float, 32 * 160 * 160 * sizeof(float));
	half2float<<<GRID(32 * 160 * 160), BLOCK>>>(output_mask, mask_float, 32 * 160 * 160);
	hipDeviceSynchronize();
	
	float* scores;
	int  * ids;
	hipMalloc(&scores, 8400 * sizeof(float));
	hipMalloc(&ids, 8400 * sizeof(int));
	
	int* size_dev;
	hipMalloc(&size_dev, sizeof(int));
	hipMemset(size_dev, 0, sizeof(int));
	find_max_score_kernel<<<GRID(8400), BLOCK>>>(box_data, scores, ids,
	                                             num_class, threshold, size_dev);
	hipDeviceSynchronize();
	
	int size;
	hipMemcpy(&size, size_dev, sizeof(int), hipMemcpyDeviceToHost);
	
	if (size)
	{
		cv::Rect* boxes_dev;
		float   * predict_mask;
		int     * class_ids_dev;
		int     * confirmed_idx;
		hipMalloc(&boxes_dev, size * sizeof(cv::Rect));
		hipMalloc(&predict_mask, size * 32 * sizeof(float));
		hipMalloc(&class_ids_dev, size * sizeof(int));
		hipMalloc(&confirmed_idx, size * sizeof(int));
		
		float x_factor = (float) image_size.width / (float) input_size.width;
		float y_factor = (float) image_size.height / (float) input_size.height;
		
		score_check_kernel<<<1, 1>>>(ids, scores, class_ids_dev, confirmed_idx);
		predict_kernel<<<GRID(size), BLOCK>>>(confirmed_idx, box_data, predict_mask, boxes_dev, size, num_box_rows,
		                                      x_factor, y_factor, image_size);
		hipDeviceSynchronize();
		
		int* class_ids = new int[size];
		hipMemcpy(class_ids, class_ids_dev, size * sizeof(int), hipMemcpyDeviceToHost);
		
		auto masks_maps = mask_process(predict_mask, mask_float, boxes_dev, size);
		
		auto boxes = new cv::Rect[size];
		hipMemcpy(boxes, boxes_dev, size * sizeof(cv::Rect), hipMemcpyDeviceToHost);
		
		mask_img = drawMasks(_input_img, boxes, class_ids, size, 0.3, masks_maps);
		
		delete[] boxes;
		delete[] class_ids;
		
		hipFree(confirmed_idx);
		hipFree(class_ids_dev);
		hipFree(predict_mask);
		hipFree(boxes_dev);
	}
	
	hipFree(output_box);
	hipFree(output_mask);
	hipFree(size_dev);
	hipFree(ids);
	hipFree(scores);
	hipFree(mask_float);
	
	return mask_img;
}

std::vector<cv::cuda::GpuMat> YoloSeg::mask_process(float* _pred_mask,
                                                    float* _output_mask,
                                                    const cv::Rect* _boxes_dev,
                                                    int _size) const
{
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	float alpha = 1.f;
	float beta  = 0.f;
	
	float* masks_dev;
	hipMalloc(&masks_dev, _size * 160 * 160 * sizeof(float));
	hipMemset(masks_dev, 0, _size * 160 * 160 * sizeof(float));
	
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 2, 2, 2, &alpha, _pred_mask, _size, _output_mask, 32, &beta,
	               masks_dev, _size);
	
	hipblasDestroy(handle);
	
	std::vector<cv::cuda::GpuMat> masks_vec(_size);
	
	for (int i = 0; i < _size; i++)
		masks_vec[ i ] = cv::cuda::GpuMat(160, 160, CV_32F, masks_dev + 160 * 160 * i);
	
	std::vector<cv::cuda::GpuMat> mask_maps(_size);
	
	float x_factor = 160.f / (float) image_size.width;
	float y_factor = 160.f / (float) image_size.height;
	
	cv::Size blur_size(static_cast<int>(1.f / x_factor), static_cast<int>(1.f / y_factor));
	if (blur_size.width % 2 == 0) blur_size.width++;
	if (blur_size.height % 2 == 0) blur_size.height++;
	
	cv::Ptr<cv::cuda::Filter> blur_filter = cv::cuda::createGaussianFilter(CV_32F, CV_32F, blur_size, 0);
	
	auto* boxes = (cv::Rect*) malloc(_size * sizeof(cv::Rect));
	hipMemcpy(boxes, _boxes_dev, _size * sizeof(cv::Rect), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < _size; i++)
	{
		cv::Mat          mask_cpu;
		cv::cuda::GpuMat mask_map;
		cv::Rect         box;
		
		mask_cpu = cv::Mat::zeros(image_size.height, image_size.width, CV_8U);
		mask_map.upload(mask_cpu);
		
		box.x      = static_cast<int>((float) boxes[ i ].x * x_factor);
		box.y      = static_cast<int>((float) boxes[ i ].y * x_factor);
		box.width  = static_cast<int>((float) boxes[ i ].width * y_factor);
		box.height = static_cast<int>((float) boxes[ i ].height * y_factor);
		
		if (box.x + box.width > 160) box.width   = 160 - box.x;
		if (box.y + box.height > 160) box.height = 160 - box.y;
		
		cv::cuda::GpuMat scale_crop_mask = masks_vec[ i ](box);
		cv::cuda::GpuMat crop_mask;
		cv::cuda::resize(scale_crop_mask, crop_mask, box.size(), 0, 0, cv::INTER_CUBIC);
		scale_crop_mask.release();
		
		blur_filter->apply(crop_mask, crop_mask);
		
		cv::cuda::GpuMat temp;
		cv::cuda::compare(crop_mask, 0.5, temp, cv::CMP_GT);
		
		temp.copyTo(mask_map(box));
		temp.release();
		
		mask_map.copyTo(mask_maps[ i ]);
		mask_map.release();
	}
	
	delete[] boxes;
	hipFree(masks_dev);
	
	return mask_maps;
}

cv::Mat YoloSeg::drawMasks(const cv::Mat &_image,
                           cv::Rect* _boxes,
                           const int* _class_ids,
                           int _size,
                           double _mask_alpha,
                           std::vector<cv::cuda::GpuMat> _mask_maps)
{
	cv::Mat mask_img;
	_image.convertTo(mask_img, mask_img.type());
	
	for (int i = 0; i < _size; i++)
	{
		cv::Rect box      = _boxes[ i ];
		int      class_id = _class_ids[ i ];
		
		cv::Scalar color = cv::Scalar(1, 0, 0);
		
		if (_mask_maps.empty())
		{
			cv::rectangle(mask_img, box, color, -1);
		}
		else
		{
			cv::Mat mask_map;
			_mask_maps[ i ].download(mask_map);
			cv::Mat subImg = mask_map(box);
			
			cv::Mat subImg32F;
			subImg.convertTo(subImg32F, CV_32FC3);
			cv::cvtColor(subImg32F, subImg32F, cv::COLOR_GRAY2RGB);
			
			cv::Mat colorMat(subImg.rows, subImg.cols, CV_32FC3, color);
			
			cv::cuda::multiply(subImg32F, ( 1 - 0.0 ), subImg32F);
			cv::cuda::addWeighted(subImg32F, 1.0, colorMat, 0.0, 0, subImg32F);
			
			cv::Mat subImg8U;
			subImg32F.convertTo(subImg8U, CV_8UC3);
			mask_img(box).copyTo(subImg8U);
		}
	}
	
	cv::Mat result;
	cv::cuda::addWeighted(mask_img, 1.0, _image, 1 - 0.0, 0, result);
	
	return result;
}